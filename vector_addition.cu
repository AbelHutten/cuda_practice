#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vecAddKernel(float* A_d, float* B_d, float* C_d, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C_d[i] = A_d[i] + B_d[i];
    }
}

// Function to add two vectors using CUDA
void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    float *A_d, *B_d, *C_d;  // Device pointers

    // Allocate memory on the GPU
    hipMalloc((void**)&A_d, n * sizeof(float));
    hipMalloc((void**)&B_d, n * sizeof(float));
    hipMalloc((void**)&C_d, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(A_d, A_h, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, n * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch kernel
    vecAddKernel<<<gridSize, blockSize>>>(A_d, B_d, C_d, n);

    // Copy result from device to host
    hipMemcpy(C_h, C_d, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

// Function to print a vector
void printVec(float* A, int N) {
    for (int i = 0; i < N; i++) {
        printf("%2d ", (int)floor(A[i]));
    }
    printf("\n");
}

// Function to create a dash separator
char* createDashString(int n) {
    char *str = (char*) malloc(n + 2);
    if (str == NULL) {
        perror("Memory allocation failed");
        exit(1);
    }
    memset(str, '-', n);
    str[n] = '\n';
    str[n+1] = '\0';
    return str;
}

int main() {
    int N = 40;  // Size of vectors

    // Allocate memory for vectors on the host (CPU)
    float* A = (float*)malloc(N * sizeof(float));
    float* B = (float*)malloc(N * sizeof(float));
    float* C = (float*)malloc(N * sizeof(float));

    if (A == NULL || B == NULL || C == NULL) {
        printf("Memory allocation failed\n");
        return 1;
    }

    // Initialize vectors with random values
    for (int i = 0; i < N; i++) {
        A[i] = (float)(rand() % 50);
        B[i] = (float)(rand() % 50);
    }

    // Perform vector addition on the GPU
    vecAdd(A, B, C, N);

    // Wait for kernel to finish before continuing
    hipDeviceSynchronize();


    // Print results
    printVec(A, N);
    printVec(B, N);
    printf("%s", createDashString(3 * N - 1));
    printVec(C, N);

    // Free allocated memory
    free(A);
    free(B);
    free(C);

    return 0;
}
